#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <iostream>
#include <ctime>
#include "../ImageLoader/ImageLoader.h"

__global__ void changeColors(Pixel* pixel_dev, int brightness)
{
	//pixel_dev[blockIdx.x].r = 166;
	//int index = blockIdx.x + threadIdx.x;
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;
	int r = pixel_dev[offset].r + brightness;
	int g = pixel_dev[offset].g + brightness;
	int b = pixel_dev[offset].b + brightness;

	if (r < 0)
		r = 0;
	if (g < 0)
		g = 0;
	if (b < 0)
		b = 0;

	if (r > 255)
		r = 255;
	if (g > 255)
		g = 255;
	if (b > 255)
		b = 255;

	pixel_dev[offset].r = r;
	pixel_dev[offset].g = g;
	pixel_dev[offset].b = b;
}

__global__ void horizontalGaussianBlur(Pixel* pixel_dev)
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;
	static const float weight[5] = {0.2270270270, 0.1945945946, 0.1216216216, 0.0540540541, 0.0162162162};
	int colorCanal = 0;

	colorCanal = pixel_dev[offset].r * weight[0];
	for(int i = 1; i < 5; i++)
	{
		colorCanal += pixel_dev[offset + i].r * weight[i];
		colorCanal += pixel_dev[offset - i].r * weight[i];
	}
	pixel_dev[offset].r = colorCanal;

	colorCanal = pixel_dev[offset].g * weight[0];
	for (int i = 1; i < 5; i++)
	{
		colorCanal += pixel_dev[offset + i].g * weight[i];
		colorCanal += pixel_dev[offset - i].g * weight[i];
	}
	pixel_dev[offset].g = colorCanal;


	colorCanal = pixel_dev[offset].b * weight[0];
	for (int i = 1; i < 5; i++)
	{
		colorCanal += pixel_dev[offset + i].b * weight[i];
		colorCanal += pixel_dev[offset - i].b * weight[i];
	}
	pixel_dev[offset].b = colorCanal;
}

__global__ void verticalGaussianBlur(Pixel* pixel_dev)
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;
	//static int executed = 0;
	static const float weight[5] = { 0.2270270270, 0.1945945946, 0.1216216216, 0.0540540541, 0.0162162162 };
	int colorCanal = 0;
	//printf("\n Block %d Offset %d Executed %d", blockIdx.x, offset, executed);
	colorCanal = pixel_dev[offset].r * weight[0];
	for (int i = 1; i < 5; i++)
	{
		int index = offset + i * gridDim.x;
		if(index < gridDim.x * gridDim.y)
		{
			colorCanal += pixel_dev[offset + i * gridDim.x].r * weight[i];
		}
		index = offset - i * gridDim.x;
		if (index >= 0)
		{
			colorCanal += pixel_dev[offset - i * gridDim.x].r * weight[i];
		}
	}
	pixel_dev[offset].r = colorCanal;

	colorCanal = pixel_dev[offset].g * weight[0];
	for (int i = 1; i < 5; i++)
	{
		int index = offset + i * gridDim.x;
		if (index < gridDim.x * gridDim.y)
		{
			colorCanal += pixel_dev[offset + i * gridDim.x].g * weight[i];
		}
		index = offset - i * gridDim.x;
		if (index >= 0)
		{
			colorCanal += pixel_dev[offset - i * gridDim.x].g * weight[i];
		}
	}
	pixel_dev[offset].g = colorCanal;


	colorCanal = pixel_dev[offset].b * weight[0];
	for (int i = 1; i < 5; i++)
	{
		int index = offset + i * gridDim.x;
		if (index < gridDim.x * gridDim.y)
		{
			colorCanal += pixel_dev[offset + i * gridDim.x].b * weight[i];
		}
		index = offset - i * gridDim.x;
		if (index >= 0)
		{
			colorCanal += pixel_dev[offset - i * gridDim.x].b * weight[i];
		}
	}
	//executed++;
	pixel_dev[offset].b = colorCanal;
	
}

void runKernel(int iterations)
{
	std::clock_t start;
	double duration;
	start = std::clock();

	ImageLoader* imgLoader = ImageLoader::getInstance();
	int width = imgLoader->width, height = imgLoader->height, channels = imgLoader->channels;
	int size = width * height;
	Pixel *pixels = ImageLoader::getInstance()->pixels;

	Pixel *pixels_dev;
	
	hipMalloc(&pixels_dev, size * sizeof(Pixel));
	hipMemcpy(pixels_dev, pixels, size * sizeof(Pixel), hipMemcpyHostToDevice);

	dim3 grid(width, height);
	
	for (int i = 0; i < iterations; i++)
	{
		horizontalGaussianBlur <<<grid, 1 >>> (pixels_dev);
		verticalGaussianBlur<<<grid,1>>>(pixels_dev);
		changeColors <<<grid, 1>>> (pixels_dev, 12);
		hipGetLastError();
	}
	hipMemcpy(pixels, pixels_dev, size * sizeof(Pixel), hipMemcpyDeviceToHost);
	hipFree(pixels_dev);

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	printf("Duration %f", duration);
	
	imgLoader->writeImage();
}
